
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <iostream>
#include <cmath>
#include <functional>
#include <ctime>
#include <chrono>




using namespace std;

//function to round the result to two decimal places
double apr(double x){
  double d=round(x*100);
  x=d/100;
  return x;
}

//a function that calculates the integral of a polynomial function of the form f(x)=a*x^n+b*x^(n-1)+...+y*x+z

__global__ void integral(double x_1, double x_2, double d,double *results){
  
  const int s=2; //polynomial degree
  double w[s+1]={1,1,1}; //array of polynomial coefficients {a,b,c,...}
  double y1=0, y2=0;

  int N=(x_2-x_1)/d; //number of steps
  int t = blockIdx.x * blockDim.x + threadIdx.x; //variable to check if we are not going out of range
  if (t<N)
  {
   for (int i=0; i<s+1; i++){
       y1+=pow(x_1+d*t,i)*w[s-i]; 
       y2+=pow(x_1+d*(t+1),i)*w[s-i]; 
   }
   results[t]=(y1+y2)*0.5*d; //area under a trapezoid with bases y1 and y2 and height d
  }
}

int main() {
  
  double x_1=-5.,x_2=5.;//integration limits
  double d=0.001; //length of step
  int N=(x_2-x_1)/d; //number of steps

  
  double *result; //array of results
  double r=0.; //result
  size_t size = N*sizeof(double);
  hipMallocManaged(&result, size);

  int threads_per_block = 256;
  int number_of_blocks = (N / threads_per_block) + 1;

  //execution time
  
  //mirzenie czasu juhuu
  const int n=1000;
  double t_time[n]={0};
  double mean=0;

  chrono::steady_clock::time_point s_r1;
  chrono::steady_clock::time_point e_r1;
  
  int m=0;
  for (int i=0; i<n; i++){
    //execution time
    s_r1 = chrono::steady_clock::now();
  
    integral<<< number_of_blocks, threads_per_block >>>(x_1, x_2, d, result);
    hipDeviceSynchronize();
    
    e_r1 = chrono::steady_clock::now();

    t_time[i]=chrono::duration_cast<chrono::microseconds> (e_r1 - s_r1).count();
    // wyswietlanie otrzymanych czasow
    cout<<i+1<<". "<<t_time[i]<<endl;
    //filtrowanie anomalnie du¿ych wyników
    if (t_time[i]<500){
      mean+=t_time[i];
      m++;
    }
  }

  mean=mean/m;

  cout<<"Mean: "<<mean<<endl;

  for (int i=0; i<N; i++){
      r+=result[i];
  }
  r=apr(r);
   
  
  //result
  

  cout<<"Integral of a polynomial function in limits ("<<x_1<<","<<x_2<<") equals = "<<r<<endl;
  //cout<<"Execution time: "<< chrono::duration_cast<chrono::microseconds> (e_r1 - s_r1).count() << " us"<<endl<<endl;
 

  hipFree(result);
}
